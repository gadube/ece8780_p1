#include "hip/hip_runtime.h"
#include "im2Gray.h"


#ifndef BLOCK
#define BLOCK 32
#endif

#ifndef TILE_WIDTH
#define TILE_WIDTH 4
#endif

/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */
	size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x < numCols && y < numRows)
	{
		// gray pixel index
		int i = y * numCols + x;

		unsigned char r = d_in[i].x; // red pixel value
		unsigned char g = d_in[i].y; // green pixel value
		unsigned char b = d_in[i].z; // blue pixel value

		// grayscale conversion using formula 1 from project doc
		d_grey[i] = 0.299f * r + 0.587f * g + 0.114f * b;
	}
	return;
}


__global__ 
void im2Gray_s(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){

	__shared__ uchar4 ds_in[TILE_WIDTH][TILE_WIDTH];
	__shared__ unsigned char ds_grey[TILE_WIDTH][TILE_WIDTH];

	size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	int i = y * numCols + x;

	for(int p =0; p < numCols/TILE_WIDTH; ++p){
		if (x < numCols && y < numRows) {
			ds_in[threadIdx.y][threadIdx.x] =  d_in[y*numCols + p*TILE_WIDTH+threadIdx.x];
		}
		__syncthreads();

		//for(int j = 0; j < TILE_WIDTH; j++){
			if (x < numCols && y < numRows){

				// gray pixel index
				unsigned char r = ds_in[threadIdx.y][threadIdx.x].x; // red pixel value
				unsigned char g = ds_in[threadIdx.y][threadIdx.x].y; // green pixel value
				unsigned char b = ds_in[threadIdx.y][threadIdx.x].z; // blue pixel value

				// grayscale conversion using formula 1 from project doc
				ds_grey[threadIdx.y][threadIdx.x] = 0.299f*r + 0.587f * g + 0.114f * b;
			//}

		}
		__syncthreads();
		/*if (x < numCols && y < numRows) {
			d_grey[i] = ds_grey[threadIdx.y][threadIdx.x];
		}*/
	}
	return;
}


void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // configure launch params here 


    dim3 block(BLOCK,BLOCK,1);
    dim3 grid((numCols + BLOCK - 1)/BLOCK,(numRows + BLOCK - 1)/BLOCK,1);

    im2Gray_s<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    
}





