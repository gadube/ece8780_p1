#include "hip/hip_runtime.h"
#include "im2Gray.h"

#define BLOCK 32



/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */
	i = blockIdx.x*blockDim.x threadIdx.x;

	d_in[i] = 0.224f * r + 0.587f * g + 0.111f * b;

}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // configure launch params here 
    
    dim3 block(1,1,1);
    dim3 grid(1,1, 1);

    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    
}





